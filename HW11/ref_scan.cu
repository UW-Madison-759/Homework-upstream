// Reference Scan implementation - Author: Ananoymous student of ME759 Fall 2017
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>


int checkResults(float*res, float* cudaRes,int length)
{
	int nDiffs=0;
	const float smallVal = .3f; // Keeping this extra high as we have repetitive addition and sequence matters
	for(int i=0; i<length; i++)
		if(fabs(cudaRes[i]-res[i])>smallVal)
			{nDiffs++;
       //printf("%f %f\n",cudaRes[i],res[i]);
      }
	return nDiffs;
}

void initializeArray(FILE* fp,float* arr, int nElements)
{
	for( int i=0; i<nElements; i++){
		int r=fscanf(fp,"%f",&arr[i]);
		if(r == EOF){
			rewind(fp);
		}
		arr[i]-=5; // This is to make the data zero mean. Otherwise we reach large numbers and lose precision
	}
}

void inclusiveScan_SEQ(float *in, float *out,int length) {
	float sum=0.f;
	for (int i =0; i < length; i++) {
		sum+=in[i];
		out[i]=sum;
	}
}

__global__ void scan(float *g_odata, float *g_idata,int n){
  extern volatile __shared__ float temp[];
  int thid = threadIdx.x;
  int pout = 0,pin = 1;
  
  if(thid<n)
    temp[thid] = g_idata[thid];
  else
    temp[thid] = 0.0f;
  __syncthreads();
  
  for(int offset = 1;offset<n;offset <<=1)
  {
    pout = 1- pout;
    pin = 1 - pout;
    
    if(thid >= offset)
      temp[pout*n+thid] = temp[pin*n+thid]+temp[pin*n+thid-offset];
    else
      temp[pout*n+thid] = temp[pin*n+thid];
      
    __syncthreads();
  }
  if(thid<n)
    g_odata[thid] = temp[pout*n+thid];
}

__global__ void scanlarge(float *g_odata, float *g_idata,float *aux_data,int n,int arraysize){
  extern volatile __shared__ float temp[];
  int thid = threadIdx.x;
  int start = blockIdx.x*1024;
  int aux_in = blockIdx.x;
  int pout = 0,pin = 1;
  
  if(thid+start<arraysize)
    temp[thid] = g_idata[thid+start];
  else
    temp[thid] = 0.00;
  __syncthreads();
  
  for(int offset = 1;offset<n;offset <<=1)
  {
    pout = 1- pout;
    pin = 1 - pout;
    
    if(thid >= offset)
      temp[pout*n+thid] = temp[pin*n+thid]+temp[pin*n+thid-offset];
    else
      temp[pout*n+thid] = temp[pin*n+thid];
      
    __syncthreads();
  }
  if(thid+start<arraysize){
    g_odata[thid+start] = temp[pout*n+thid];}
  aux_data[aux_in] = temp[1023];
}

__global__ void addscan(float *g_odata,float *g_idata,float *aux_data,int arraysize){
  extern volatile __shared__ float temp[];
  int thid = threadIdx.x;
  int start = (blockIdx.x+1)*1024;
  int aux_in = blockIdx.x;
  
  if(thid+start<arraysize)
    temp[thid] = g_idata[thid+start];
  else
    temp[thid] = 0.00;
  __syncthreads();
  
  temp[thid]+=aux_data[aux_in];
  __syncthreads();  
  if(thid+start<arraysize)
    g_odata[thid+start]=temp[thid];
}


int main(int argc, char* argv[]) {
	if(argc!=2){
		printf("Usage %s N\n",argv[0]);
		return 1;
	}
	int N=atoi(argv[1]);
	FILE *fp = fopen("problem1.inp","r");
	int size = N * sizeof(float); 
	//allocate resources
  int threadsperblock,blocksPerGrid;
	float *in      = (float *)malloc(size);
	float *out     = (float *)malloc(size); 
	float *cuda_out= (float *)malloc(size);
	float time = 0.f;
	initializeArray(fp,in, N);
	// Your code here
 
  float *dout,*din;
  
  hipMalloc((void**)&dout,size);
	hipMalloc((void**)&din,size);
 
  hipEvent_t startEvent_inc, stopEvent_inc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
  hipEventRecord(startEvent_inc,0); // starting timing for inclusive  
  
  hipMemcpy(din,in,size,hipMemcpyHostToDevice);
  hipMemset(dout,0,size);
  
  if(N>1024)
  {
     threadsperblock = 1024;
     blocksPerGrid = (N+threadsperblock-1)/threadsperblock;
     float *aux;
     float *auxscan;
     int num = 1024;
     hipMalloc((void**)&aux,sizeof(float)*blocksPerGrid);
     hipMemset(aux,0,sizeof(float)*blocksPerGrid);
     hipMalloc((void**)&auxscan,sizeof(float)*blocksPerGrid);
     hipMemset(auxscan,0,sizeof(float)*blocksPerGrid);
     
     if(blocksPerGrid<=1024)
     {
       scanlarge<<<blocksPerGrid,threadsperblock,2048*sizeof(float)>>>(dout,din,aux,num,N);
       hipDeviceSynchronize();
       // Scanning the auxilliary array
       scan<<<1,blocksPerGrid,2*blocksPerGrid*sizeof(float)>>>(auxscan,aux,blocksPerGrid);
       hipDeviceSynchronize();
       // Adding the scanned block to get final result
       addscan<<<blocksPerGrid,threadsperblock,2048*sizeof(float)>>>(dout,dout,auxscan,N);
       hipDeviceSynchronize();
       hipMemcpy(cuda_out,dout,size,hipMemcpyDeviceToHost);
     }
     else
     {
       float *auxblock;
       float *auxscanblock;
       float *auxout;
       volatile int blocksperGridaux = (blocksPerGrid+1023)/1024;
       //int blocksperGridaux = 2;
       hipMalloc((void**)&auxblock,sizeof(float)*blocksperGridaux);
       hipMemset(auxblock,0,sizeof(float)*blocksperGridaux);
       hipMalloc((void**)&auxscanblock,sizeof(float)*blocksperGridaux);
       hipMemset(auxscanblock,0,sizeof(float)*blocksperGridaux);
       hipMalloc((void**)&auxout,sizeof(float)*blocksPerGrid);
       hipMemset(auxout,0,sizeof(float)*blocksPerGrid);
       
       scanlarge<<<blocksPerGrid,threadsperblock,2048*sizeof(float)>>>(dout,din,aux,num,N); // We get the block sums here
       hipDeviceSynchronize();
       
       
       scanlarge<<<blocksperGridaux,threadsperblock,2048*sizeof(float)>>>(auxout,aux,auxblock,num,blocksPerGrid); // Block sum array size is greater than 1024. So repeat the whole > 1024 process
       hipDeviceSynchronize();
       scan<<<1,blocksperGridaux,2*blocksperGridaux*sizeof(float)>>>(auxscanblock,auxblock,blocksperGridaux); // Aux sum of sux array
       hipDeviceSynchronize();
       addscan<<<blocksperGridaux-1,threadsperblock,2048*sizeof(float)>>>(auxout,auxout,auxscanblock,blocksPerGrid); // Fully scanned auxilliary array
       hipDeviceSynchronize();
       
       
       addscan<<<blocksPerGrid,threadsperblock,2048*sizeof(float)>>>(dout,dout,auxout,N);
       hipDeviceSynchronize();
       hipMemcpy(cuda_out,dout,size,hipMemcpyDeviceToHost);
     }
  }
  else{
     threadsperblock = N;
     blocksPerGrid = 1; 
     scan<<<blocksPerGrid,threadsperblock,2*size>>>(dout,din,N);
     hipMemcpy(cuda_out,dout,size,hipMemcpyDeviceToHost);
     } 
  hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
  hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&time, startEvent_inc, stopEvent_inc);   
 
	inclusiveScan_SEQ(in, out,N);
	int nDiffs = checkResults(out, cuda_out,N);

	if(nDiffs)printf("Test Failed\n"); // This should never print
	printf("%d\n%f\n%f\n",N,cuda_out[N-1],time);
  //printf("%d\n",nDiffs);
  //printf("%f\n",out[N-1]);


	//free resources 
	free(in); free(out); free(cuda_out);
  hipFree(din);hipFree(dout);
  //cudaFree(dsize);
	return 0;
}
