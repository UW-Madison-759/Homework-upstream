#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(value, label) {              \
   hipError_t c = (value);                     \
   if (c != hipSuccess) {                      \
   fprintf(stderr,                              \
     "Error: '%s' at line %d in %s\n",          \
     hipGetErrorString(c),__LINE__,__FILE__);  \
   goto label;                                  \
   } }

static __global__ void prefix_scan_device(float *in, float *out, int size) {
	// Do CUDA stuff
}

void prefix_scan(float *in, float *out, int size) {
	float *d_in=0, *d_out=0;
	CUDA_CHECK(hipMalloc(&d_in, size * sizeof(float)), cuda_error)
	CUDA_CHECK(hipMalloc(&d_out, size * sizeof(float)), cuda_error)
	
	CUDA_CHECK(hipMemcpy(d_in, in, size * sizeof(float), hipMemcpyHostToDevice), cuda_error)
	prefix_scan_device<<<128, 1>>>(d_in, d_out, size);
	CUDA_CHECK(hipMemcpy(out, d_out, size * sizeof(float), hipMemcpyDeviceToHost), cuda_error)

cuda_error:
	if(d_in) hipFree(d_in);
	if(d_out) hipFree(d_out);
}
