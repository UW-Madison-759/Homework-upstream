#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void sum(int *x) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	x[index] = blockIdx.x + threadIdx.x;
}

int main() {	
	const int N = 16;
	int x[N];
	int *dArray;
	hipMalloc((void**) &dArray, sizeof(int) * N);
	
	sum<<<2,8>>>(dArray);
	hipMemcpy(x, dArray, sizeof(int) * N, hipMemcpyDeviceToHost);	
	
	for(int i=0; i<N; i++)
		printf("%d\n", x[i]);
	
	hipFree(dArray);
	return 0;
}
