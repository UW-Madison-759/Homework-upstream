#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>
int main( int argc, char *argv[])
{

	if(argc!=3)
	{
		printf("Invalid argument Usage: ./problem3 N M");
		return 0;
	}

	FILE *fpA,*fpB;
	int N = atoi(argv[1]);
	int M = atoi(argv[2]); 
	double *hA= new double[N];
	double *hB= new double[N];
	double *hC=  new double[N];
	double *refC=  new double[N]; // Used to verify functional correctness
	double *dA,*dB,*dC;  // You may use these to allocate memory on gpu
	//defining variables for timing
	hipEvent_t startEvent_inc, stopEvent_inc, startEvent_exc, stopEvent_exc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
	hipEventCreate(&startEvent_exc);
	hipEventCreate(&stopEvent_exc);
	float elapsedTime_inc, elapsedTime_exc;

	//reading files
	fpA = fopen("inputA.inp", "r");
	fpB= fopen("inputB.inp", "r");


	for (int i=0;i<N;i++){    
		fscanf(fpA, "%lf",&hA[i]);
	}
	for (int i=0;i<N;i++){
		fscanf(fpB, "%lf",&hB[i]);
	}



      for(int i=0;i<N;i++)
        refC[i]=hA[i]+hB[i];


	hipEventRecord(startEvent_inc,0); // starting timing for inclusive
	// TODO allocate memory for arrays and copay array A and B

	hipEventRecord(startEvent_exc,0); // staring timing for exclusive

	// TODO launch kernel 

	hipEventRecord(stopEvent_exc,0);  // ending timing for exclusive
	hipEventSynchronize(stopEvent_exc);   
	hipEventElapsedTime(&elapsedTime_exc, startEvent_exc, stopEvent_exc);

	// TODO copy data back


	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
	hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&elapsedTime_inc, startEvent_inc, stopEvent_inc);



	//verification
	int count=0;
	for(int i=0;i<N;i++)
	{
		if(hC[i]!=refC[i])
		{
			count++;
		}
	}
	if(count!=0) // This should never be printed in correct code
		std::cout<<"Error at "<< count<<" locations\n";
	std::cout<<N<<"\n"<<M<<"\n"<<elapsedTime_exc<<"\n"<<elapsedTime_inc<<"\n"<<hC[N-1]<<"\n";
	//freeing memory
	delete[] hA,hB,hC,refC;     

	// TODO free CUDA memory allocated

	return 0;
}
