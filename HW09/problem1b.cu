#include<iostream>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define RADIUS 3

int checkResults(int startElem, int endElem, float* cudaRes, float* res)
{
    int nDiffs=0;
    const float smallVal = 0.0001f;
    for(int i=startElem; i<endElem; i++)
        if(fabs(cudaRes[i]-res[i])>smallVal)
            nDiffs++;
    return nDiffs;
}

void initializeWeights(float* weights, int rad)
{
    // for now hardcoded for RADIUS=3
    weights[0] = 0.50f;
    weights[1] = 0.75f;
    weights[2] = 1.25f;
    weights[3] = 2.00f;
    weights[4] = 1.25f;
    weights[5] = 0.75f;
    weights[6] = 0.50f;
}
void initializeArray(FILE* fp,float* arr, int nElements)
{
    for( int i=0; i<nElements; i++){
                fscanf(fp,"%f",&arr[i]);
                if(getc(fp) == EOF) rewind(fp);
    }
}

void applyStencil1D_SEQ(int sIdx, int eIdx, const float *weights, float *in, float *out) {
  
  for (int i = sIdx; i < eIdx; i++) {   
    out[i] = 0;
    //loop over all elements in the stencil
    for (int j = -RADIUS; j <= RADIUS; j++) {
      out[i] += weights[j + RADIUS] * in[i + j]; 
    }
    out[i] = out[i] / (2 * RADIUS + 1);
  }
}

__global__ void applyStencil1D(int sIdx, int eIdx, const float *weights, float *in, float *out) {
    int i = sIdx + blockIdx.x*blockDim.x + threadIdx.x;
    if( i < eIdx ) {
        float result = 0.f;
        result += weights[0]*in[i-3];
        result += weights[1]*in[i-2];
        result += weights[2]*in[i-1];
        result += weights[3]*in[i];
        result += weights[4]*in[i+1];
        result += weights[5]*in[i+2];
        result += weights[6]*in[i+3];
        result /=7.f;
        out[i] = result;
    }
}

int main(int argc, char* argv[]) {
  if(argc!=2){
        printf("Usage %s N\n",argv[0]);
	return 1;
  }
  int N=atoi(argv[1]);
  FILE *fp = fopen("problem1.inp","r");
  int size = N * sizeof(float); 
  int wsize = (2 * RADIUS + 1) * sizeof(float); 
  //allocate resources
  float *weights, *in, *cuda_out; 
  hipHostMalloc((void **)&weights, wsize, hipHostMallocDefault);
  hipHostMalloc((void **)&in, size, hipHostMallocDefault); 
  hipHostMalloc((void **)&cuda_out, size, hipHostMallocDefault); 

  float *out     = (float *)malloc(size); 
  float time = 0.f;
  initializeWeights(weights, RADIUS);
  initializeArray(fp,in, N);
  float *d_weights;  hipMalloc(&d_weights, wsize);
  float *d_in;       hipMalloc(&d_in, size);
  float *d_out;      hipMalloc(&d_out, size);
  
  hipMemcpy(d_weights,weights,wsize,hipMemcpyHostToDevice);
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
  applyStencil1D<<<(N+511)/512, 512>>>(RADIUS, N-RADIUS, d_weights, d_in, d_out);
  hipMemcpy(cuda_out, d_out, size, hipMemcpyDeviceToHost);

  applyStencil1D_SEQ(RADIUS, N-RADIUS, weights, in, out);
  int nDiffs = checkResults(RADIUS, N-RADIUS, cuda_out, out);
  if(nDiffs)printf("Test Failed\n"); // This should never print
  printf("%f\n%f\n",cuda_out[N-RADIUS-1],time);
  //free resources 
  hipHostFree(weights); hipHostFree(in); hipHostFree(cuda_out); 
  free(out);
  hipFree(d_weights);  hipFree(d_in);  hipFree(d_out);
  return 0;
}
