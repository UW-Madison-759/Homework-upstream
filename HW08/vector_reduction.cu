#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project

// includes, kernels
#include "vector_reduction_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward

float* read_array(const char* filename, int len) {
	float *x = (float*) malloc(len * sizeof(float));
	FILE *fp = fopen(filename, "r");
	for (int i = 0; i < len; i++) {
		fscanf(fp, "%f", &x[i]);
	}
	fclose(fp);
	return x;
}

float computeOnDevice(float* h_data, int num_elements);

extern "C" void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
//! Run test
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	int num_elements = NUM_ELEMENTS;

	float* h_data=read_array("problem2.inp",num_elements);

	// * No arguments: Randomly generate input data and compare against the 
	//   host's result.
	// * One argument: Read the input data array from the given file.
	// compute reference solution
	float reference = 1.0f;  
	computeGold(&reference , h_data, num_elements);

	// **===-------- Modify the body of this function -----------===**
	float result = computeOnDevice(h_data, num_elements);
	// **===-----------------------------------------------------------===**


	// Run accuracy test
	float epsilon = 0.0001f;
	unsigned int result_regtest = (abs(result - reference) <= epsilon);

	if(!result_regtest)printf("Test failed device: %f  host: %f\n",result,reference);//This shouldnt print in working case
	printf("%f\n",result);
	// cleanup memory
	free( h_data);
	return 0;
}



// **===----------------- Modify this function ---------------------===**
// Take h_data from host, copies it to device, setup grid and thread 
// dimensions, excutes kernel function, and copy result of scan back
// to h_data.
// Note: float* h_data is both the input and the output of this function.
float computeOnDevice(float* h_data, int num_elements)
{

	// placeholder
	return 0.0f;

}

